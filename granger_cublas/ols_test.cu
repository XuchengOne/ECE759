#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h>


#define I(i, j, ld) j * ld + i

#define CUDA_CALL(x) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}} 


 #define CUBLAS_CALL(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)


int main(){

hipblasHandle_t handle;
int n = 4;
int p = 3;
int matrixSize = n*p;
float X[n * p] = { 1.0, 1.0, 3.0, -2.0,
					1.0, 2.0, -1.0, 3.0,
					2.0, 1.0, 3.0, -1.0};

float Y[n] = {1.0, 2.0, -2.0, 1.0};

  float *a, *b;
  a = (float*) malloc(sizeof(*X));
  b = (float*) malloc(sizeof(*X));
  *a = 1.0;
  *b = 0.0;

float *XtX, *XtY, *beta, *dX, *dXtX, *dXtY, *dbeta, *dY, *dXtXi;


hipblasCreate(&handle);

  XtX = (float*) malloc(p * p * sizeof(*X));
  XtY = (float*) malloc(p * sizeof(*X));
  beta = (float*) malloc(p * sizeof(*X));

  CUDA_CALL(hipMalloc((void**) &dX, n * p * sizeof(*X)));
  CUDA_CALL(hipMalloc((void**) &dXtX, p * p * sizeof(*X)));
  CUDA_CALL(hipMalloc((void**) &dXtXi, p * p * sizeof(*X)));
  CUDA_CALL(hipMalloc((void**) &dXtY, p * sizeof(*X)));
  CUDA_CALL(hipMalloc((void**) &dbeta, p * sizeof(*X)));
  CUDA_CALL(hipMalloc((void**) &dY, n * sizeof(*X)));

  CUDA_CALL(hipMemcpy(dX, X, n * p * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dY, Y, n * sizeof(float), hipMemcpyHostToDevice));

  CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, p, p, n, 
    a, dX, n, dX, n, b, dXtX, p));


 CUDA_CALL(hipMemcpy(XtX, dXtX, p * p * sizeof(float), hipMemcpyDeviceToHost));  


 //LU decomposition

int *d_pivot_array;
int *d_info_array;

int batchSize = 1;


//allocate pivoting vector and the info array

CUDA_CALL(hipMalloc((void **)&d_pivot_array, p * sizeof(int)));
CUDA_CALL(hipMalloc((void **)&d_info_array, sizeof(int)));



float **devPtrA = 0;
float **devPtrA_dev = NULL;


devPtrA =(float **)malloc(1 * sizeof(*devPtrA));
	if (devPtrA == NULL)
	{ perror("malloc"); exit(EXIT_FAILURE); }

CUDA_CALL(hipMalloc((void **) devPtrA, matrixSize * sizeof(*X)));
CUDA_CALL(hipMalloc((void **) &devPtrA_dev, 1 * sizeof(*devPtrA)));

CUDA_CALL(hipMemcpy(devPtrA_dev, devPtrA, 1 * sizeof(*devPtrA), hipMemcpyHostToDevice));
CUBLAS_CALL(hipblasSetMatrix(n, p, sizeof(a[0]), dXtX, n, devPtrA[0], n));


// // this works
// cublascall(hipblasDgetrfBatched(handle, m, devPtrA_dev,m,d_pivot_array,d_info_array,1));

// cublascall(hipblasGetMatrix(m, n, sizeof(double), devPtrA[0], m, a, m));



  CUBLAS_CALL(hipblasSgetrfBatched(handle, p, devPtrA_dev, p, d_pivot_array, d_info_array, batchSize));


//reversion

 CUBLAS_CALL(hipblasSgetriBatched(handle, p, dXtX, p, d_pivot_array, dXtXi, p, d_info_array, batchSize));
  
  CUDA_CALL(hipMemcpy(XtX, dXtXi, p * p * sizeof(float), hipMemcpyDeviceToHost));

  
  
  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, p, 1, n, 
    a, dX, n, dY, n, b, dXtY, p);

  hipblasSgemv(handle, HIPBLAS_OP_N, p, p, 
    a, dXtXi, p, dXtY, 1, b, dbeta, 1);

  CUDA_CALL(hipMemcpy(beta, dbeta, p * sizeof(float), hipMemcpyDeviceToHost));

  printf("CUBLAS matrix algebra parameter estimates:\n");
  for(i = 0; i < p; i++){
    printf("beta_%i = %0.2f\n", i, beta[i]);
  }
  printf("\n");

  hipblasDestroy(handle);

  free(X);
  free(XtX);
  free(XtY);
  free(beta);
  free(Y);
  
  CUDA_CALL(hipFree(dX));
  CUDA_CALL(hipFree(dXtX));
  CUDA_CALL(hipFree(dXtXi));
  CUDA_CALL(hipFree(dXtY));
  CUDA_CALL(hipFree(dbeta));
  CUDA_CALL(hipFree(dY));
}

