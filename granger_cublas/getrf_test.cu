#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)



void cublas_lu(int m, int n, double* a)
{
    hipblasHandle_t handle;
    double **devPtrA = 0;
    double **devPtrA_dev = NULL;
    int *d_pivot_array;
    int *d_info_array;
    int rowsA = m;
    int colsA = n;
    int matrixSizeA;
    hipError_t error;

	// allocate the pivoting vector and the info array
	cudacall(hipMalloc((void **)&d_pivot_array, n * sizeof(int)));
    cudacall(hipMalloc((void **)&d_info_array, sizeof(int)));

    cublascall(hipblasCreate(&handle));
    matrixSizeA = rowsA * colsA;

    devPtrA =(double **)malloc(1 * sizeof(*devPtrA));
 	if (devPtrA == NULL)
 	{ perror("malloc"); exit(EXIT_FAILURE); }
	
    cudacall(hipMalloc((void **) devPtrA, matrixSizeA * sizeof(double)));
    cudacall(hipMalloc((void **) &devPtrA_dev, 1 * sizeof(*devPtrA)));

    cudacall(hipMemcpy(devPtrA_dev, devPtrA, 1 * sizeof(*devPtrA), hipMemcpyHostToDevice));
    
    cublascall(hipblasSetMatrix(rowsA, colsA, sizeof(a[0]), a, rowsA, devPtrA[0], rowsA));
    // this works
    cublascall(hipblasDgetrfBatched(handle, m, devPtrA_dev,m,d_pivot_array,d_info_array,1));

    cublascall(hipblasGetMatrix(m, n, sizeof(double), devPtrA[0], m, a, m));
}


int main()
{
	const int n = 4;
	double A[n * n] = { 1.0, 1.0, 3.0, -2.0,
						1.0, 2.0, -1.0, 3.0,
						2.0, 1.0, 3.0, -1.0,
						1.0, 2.0, -2.0, 1.0 };
						
	cublas_lu(n, n, A);
		
	fprintf(stdout, "Output:\n\n");
    for(int i=0; i<n; i++)
    {
        for(int j=0; j<n; j++)
            fprintf(stdout,"%f\t",A[i*n+j]);
        fprintf(stdout,"\n");
    }					
}